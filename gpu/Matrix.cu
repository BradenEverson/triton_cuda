#include "hip/hip_runtime.h"
#include "Matrix.h"
#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_vector_types.h>
//This program completes matrix multiplication and addition on the gpu

__global__ void mul_mtx(int *a, int *b, int *c, int N, int K){
    //Get row and col for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < K){
        int tmp = 0;
        for(int i = 0; i < N; i++){
            tmp += a[row * N + i] * b[i * K + col];
        }
        c[row * N + col] = tmp;
    }
}

//Initializes matrix
void init_matrix(int *mat, int rows, int cols){
    for(int i = 0; i < rows*cols; i++){
        mat[i] = rand() % 100;
    }
}

//Verify

void verify_res(int *a, int *b, int *c, int N, int K){
    int tmp;
    for(int i = 0; i < K; i++){
        for(int j = 0; j < N; i++){
            for(int k = 0; k < N; k++){
                
            }
        }
    }
}

int main(){
    //sample size
    int N = 1 << 10;
    int K = 1 << 10;
    size_t bytes = N * N * sizeof(int);
    int *a, *b, *c;
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Initialize default matrices
    init_matrix(a, N, N);
    init_matrix(b, N, N);

    // Set CTA (coop thread array) and Grid dimensions
    int threads = 16;
    int blocks = (N + threads - 1) / threads;

    dim3 THREADS(threads, threads);
    dim3 BLOCKS(blocks, blocks);

    // Launch kernel
    mul_mtx<<<BLOCKS, THREADS>>>(a, b, c, N, N);
    hipDeviceSynchronize();
    
}
